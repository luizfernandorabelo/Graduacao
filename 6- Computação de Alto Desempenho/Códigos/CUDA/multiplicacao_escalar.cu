#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 100
#define ESCALAR 2
#define THREADS_POR_BLOCO 256

/* 
    Região de memória constante no device. É usada
    somente para leitura. Seu acesso é mais rápido
    que o da memória global e compartilhada.
*/
__device__ __constant__ int escalar_device;

__global__ void multiplicar_por_escalar(int *vetor) {
    int indice = blockIdx.x * blockDim.x + threadIdx.x;
    if (indice < TAM) vetor[indice] = vetor[indice] * escalar_device;
}

int main(void) {

    int *vetor_host, *vetor_device;

    // Inicialização de vetor no host:
    vetor_host = (int *) malloc(TAM * sizeof(int));
    for (int i = 0; i < TAM; i++)
        vetor_host[i] = i + 1;
    
    // Alocação e cópia para o device:
    hipMalloc(&vetor_device, TAM * sizeof(int));
    hipMemcpy(vetor_device, vetor_host, TAM * sizeof(int), hipMemcpyHostToDevice);

    // Cópia do escalar para a memória constante do device:
    int escalar_host = ESCALAR;
    hipMemcpyToSymbol(HIP_SYMBOL(escalar_device), &escalar_host, sizeof(int));

    int blocos_por_grade = (TAM + THREADS_POR_BLOCO - 1) / THREADS_POR_BLOCO;

    multiplicar_por_escalar<<<blocos_por_grade,THREADS_POR_BLOCO>>>(vetor_device);

    // Cópia do resultado do device para o host:
    hipMemcpy(vetor_host, vetor_device, TAM * sizeof(int), hipMemcpyDeviceToHost);

    // Impressão do resultado:
    for (int i = 0; i < TAM; i++)
        printf("%d ", vetor_host[i]);
    printf("\n");

    free(vetor_host);

    // Liberação de memória no device:
    hipFree(vetor_device);

    return 0;
}
