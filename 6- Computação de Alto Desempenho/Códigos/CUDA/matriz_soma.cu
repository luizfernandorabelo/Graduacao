#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DIMENSAO 16  // dimensão da matriz
#define THREADS 5    // quantidade de threads por dimensão  

__global__ void soma(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < DIMENSAO && j < DIMENSAO)
        c[i * DIMENSAO + j] = a[i * DIMENSAO + j] + b[i * DIMENSAO + j];
}

int main(void) {

    int *a_host, *b_host, *c_host;
    int *a_device, *b_device, *c_device;

    // Alocação das "matrizes" na CPU:
    a_host = (int *) malloc(DIMENSAO * DIMENSAO * sizeof(int));
    b_host = (int *) malloc(DIMENSAO * DIMENSAO * sizeof(int));
    c_host = (int *) malloc(DIMENSAO * DIMENSAO * sizeof(int));

    // Alocação das "matrizes" na GPU (em sua memória global):
    hipMalloc(&a_device, DIMENSAO * DIMENSAO * sizeof(int));
    hipMalloc(&b_device, DIMENSAO * DIMENSAO * sizeof(int));
    hipMalloc(&c_device, DIMENSAO * DIMENSAO * sizeof(int));

    // Preenchimento de a e b na CPU:
    for (int i = 0; i < DIMENSAO * DIMENSAO; i++) {
        a_host[i] = +i;
        b_host[i] = -i;
    }

    // Cópia de a e b para a GPU:
    hipMemcpy(a_device, a_host, DIMENSAO * DIMENSAO * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, DIMENSAO * DIMENSAO * sizeof(int), hipMemcpyHostToDevice);

    // Definição da quantidade de threads por bloco:
    dim3 threadsPorBloco(
        THREADS,  // eixo x
        THREADS   // eixo y
    );

    // Definição da quantidade de blocos por grade:
    dim3 blocosPorGrade(
        (DIMENSAO + threadsPorBloco.x - 1) / threadsPorBloco.x,  // eixo x
        (DIMENSAO + threadsPorBloco.y - 1) / threadsPorBloco.y   // eixo y
    );

    soma<<<blocosPorGrade,threadsPorBloco>>>(a_device, b_device, c_device);
        
    // Cópia de c para a CPU:
    hipMemcpy(c_host, c_device, DIMENSAO * DIMENSAO * sizeof(int), hipMemcpyDeviceToHost);

    // Impressão do resultado:
    for (int i = 0; i < DIMENSAO * DIMENSAO; i++) {
        printf("%d ", c_host[i]);
        if (i % DIMENSAO == DIMENSAO - 1) printf("\n");
    }
    printf("\n");

    free(a_host);
    free(b_host);
    free(c_host);

    // Liberação de memória da GPU:
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}
