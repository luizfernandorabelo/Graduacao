#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 10  // máximo = 512 (blocos por grade)

const int BLOCOS_POR_GRADE = 512;
const int THREADS_POR_BLOCO = 1;

__global__ void soma(int *a, int *b, int *c) {
    int indice = blockIdx.x;
    if (indice < TAM) c[indice] = a[indice] + b[indice];
}

int main(void) {

    int *a_host, *b_host, *c_host;
    int *a_device, *b_device, *c_device;

    // Alocação dos arrays da CPU:
    a_host = (int *) malloc(TAM * sizeof(int));
    b_host = (int *) malloc(TAM * sizeof(int));
    c_host = (int *) malloc(TAM * sizeof(int));

    // Alocação dos arrays da GPU:
    hipMalloc(&a_device, TAM * sizeof(int));
    hipMalloc(&b_device, TAM * sizeof(int));
    hipMalloc(&c_device, TAM * sizeof(int));

    // Preenchimento de a e b da CPU:
    for (int i = 0; i < TAM; i++) {
        a_host[i] = -i;
        b_host[i] = i * i;
    }

    // Cópia de a e b da CPU para a GPU:
    hipMemcpy(a_device, a_host, TAM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, TAM * sizeof(int), hipMemcpyHostToDevice);

    soma<<<BLOCOS_POR_GRADE,THREADS_POR_BLOCO>>>(a_device, b_device, c_device);

    // Cópia de c da GPU para a CPU:
    hipMemcpy(c_host, c_device, TAM * sizeof(int), hipMemcpyDeviceToHost);

    // Saída com o resultado:
    for (int i = 0; i < TAM; i++)
        printf("Índice %d: %d + %d = %d\n", i, a_host[i], b_host[i], c_host[i]);
    printf("\n");

    free(a_host);
    free(b_host);
    free(c_host);

    // Liberação de memória do device:
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}
