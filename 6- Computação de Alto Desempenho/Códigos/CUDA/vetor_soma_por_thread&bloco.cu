#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 10000
#define THREADS_POR_BLOCO 128

__global__ void soma(int *a, int *b, int *c) {
    int indice = blockIdx.x * blockDim.x + threadIdx.x;
    if (indice < TAM) c[indice] = a[indice] + b[indice];
}

int main(void) {

    int *a_host, *b_host, *c_host;
    int *a_device, *b_device, *c_device;

    // Alocação dos arrays da CPU:
    a_host = (int *) malloc(TAM * sizeof(int));
    b_host = (int *) malloc(TAM * sizeof(int));
    c_host = (int *) malloc(TAM * sizeof(int));

    // Alocação dos arrays da GPU:
    hipMalloc(&a_device, TAM * sizeof(int));
    hipMalloc(&b_device, TAM * sizeof(int));
    hipMalloc(&c_device, TAM * sizeof(int));

    // Preenchimento de a e b da CPU:
    for (int i = 0; i < TAM; i++) {
        a_host[i] = -i;
        b_host[i] = i * i;
    }

    // Cópia de a e b da CPU para a GPU:
    hipMemcpy(a_device, a_host, TAM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, TAM * sizeof(int), hipMemcpyHostToDevice);

    int blocos_por_grade = (TAM + THREADS_POR_BLOCO - 1) / THREADS_POR_BLOCO;

    soma<<<blocos_por_grade,THREADS_POR_BLOCO>>>(a_device, b_device, c_device);

    // Cópia de c da GPU para a CPU:
    hipMemcpy(c_host, c_device, TAM * sizeof(int), hipMemcpyDeviceToHost);

    // Saída com parte do resultado:
    for (int i = 0; i < TAM; i += 500)
        printf("Índice %d: %d + %d = %d\n", i, a_host[i], b_host[i], c_host[i]);
    printf("\n");

    free(a_host);
    free(b_host);
    free(c_host);

    // Liberação de memória do device:
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}

