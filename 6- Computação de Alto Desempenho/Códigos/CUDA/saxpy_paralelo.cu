// ŷ = a * x̂ + ŷ 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 16

__global__ void saxpy(int *x, int *y, int a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < TAM) y[i] = a * x[i] + y[i];
}

int main(void) {

    int *x_host, *y_host, *x_device, *y_device;

    // Preenchimento de x "identidade" (para facilitar teste):
    x_host = (int *) malloc(TAM * sizeof(int));
    for (int i = 0; i < TAM; i++)
        x_host[i] = i + 1;

    // Alocação e cópia para o device:
    hipMalloc(&x_device, TAM * sizeof(int));
    hipMemcpy(x_device, x_host, TAM * sizeof(int), hipMemcpyHostToDevice);

    // Preenchimento de y com zeros (para facilitar teste):
    y_host = (int *) calloc(TAM, sizeof(int));

    // Alocação e cópia para o device:
    hipMalloc(&y_device, TAM * sizeof(int));
    hipMemcpy(y_device, y_host, TAM * sizeof(int), hipMemcpyHostToDevice);

    int threads_por_bloco = 256;
    int total_blocos = (TAM + threads_por_bloco - 1) / threads_por_bloco;  // mínimo = 1 bloco

    saxpy<<<total_blocos,threads_por_bloco>>>(x_device, y_device, 2);

    // Cópia do resultado do device para o host:
    hipMemcpy(y_host, y_device, TAM * sizeof(int), hipMemcpyDeviceToHost);
    
    // Impressão do resultado:
    for (int i = 0; i < TAM; i++)
        printf("%d ", y_host[i]);
    printf("\n");

    free(x_host);
    free(y_host);

    // Liberação de memória do device:
    hipFree(x_device);
    hipFree(y_device);

    return 0;
}
