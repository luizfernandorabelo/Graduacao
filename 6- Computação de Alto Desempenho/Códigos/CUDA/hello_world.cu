#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_world(int chamada) {
    printf("Hello World da GPU da %d° chamada, bloco = %d e trhead = %d\n", chamada, blockIdx.x, threadIdx.x);
}

int main(void) {

    int blocos, threads;

    blocos = 1; threads = 10;
    hello_world<<<blocos,threads>>>(1);
    hipDeviceSynchronize();  // barreira

    blocos = 10; threads = 1;
    hello_world<<<blocos,threads>>>(2);
    hipDeviceSynchronize();  // barreira 

    printf("Hello World da CPU\n");

    hipDeviceSynchronize();  // barreira 

    return 0;
}
