#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int soma_chamada_device(int a, int b) {
    return a + b;
}

__global__ void soma_chamada_host(int a, int b, int *c) {
    *c = soma_chamada_device(a, b);
}

int main(void) {

    int a = 2, b  = 7;
    int c_host, *c_device;

    // Alocação de memória no device:
    hipMalloc(&c_device, sizeof(int));

    soma_chamada_host<<<1,1>>>(a, b, c_device);

    /*
        A função "cudaMemcpy" possui uma barreira implícita,
        sendo, então, desnecessário colocar "cudaDeviceSynchronize"
        após a chamada do kernel a ser executado no device. 
    */

    // Cópia de memória do device para a da cpu:
    hipMemcpy(&c_host, c_device, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", a, b, c_host);

    // Liberação de memória no device:
    hipFree(c_device);

    return 0;
}
